#include "hip/hip_runtime.h"
//
// Created by malyshev on 9/9/19.
//

#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "../headers/dev_array.h"
#include "../headers/block_3d_t2.h"
#include <stdlib.h>

using namespace std;

__global__ void I_block_3d_t2_kernel(int n, int k_start, int k_end, int *matrix) {
    int ROW = threadIdx.y;
    int COL = threadIdx.x;

    for (int k = k_start; k < k_end; k++) {
        matrix[ROW * n + COL] = min(matrix[ROW * n + COL], matrix[ROW * n + k] + matrix[k * n + COL]);
        __syncthreads();
    }
    return;
}

__global__ void SD_block_3d_t2_kernel(int n, int nk, int r, int *matrix) {
    int tt = blockIdx.x / 2;
    tt = tt + min(1,((tt + 1) / (nk + 1)));

    int block_row = (blockIdx.x % 2) * nk + (1 - blockIdx.x % 2) * tt;
    int block_col = (1 - blockIdx.x % 2) * nk + (blockIdx.x % 2) * tt;

    int ROW = threadIdx.y + block_row * r;
    int COL = threadIdx.x + block_col * r;

    int k_start = nk * r;
    int k_end = k_start + r;

    for (int k = k_start; k < k_end; k++) {
        matrix[ROW * n + COL] = min(matrix[ROW * n + COL], matrix[ROW * n + k] + matrix[k * n + COL]);
        __syncthreads();
    }
}

__global__ void DD_block_3d_t2_kernel(int n, int nk, int r, int *matrix) {
    int block_row = blockIdx.y + min(1,((blockIdx.y + 1) / (nk + 1)));
    int block_col = blockIdx.x + min(1,((blockIdx.x + 1) / (nk + 1)));

    int ROW = threadIdx.y + block_row * r;
    int COL = threadIdx.x + block_col * r;

    int k_start = nk * r;
    int k_end = k_start + r;

    for (int k = k_start; k < k_end; k++) {
        matrix[ROW * n + COL] = min(matrix[ROW * n + COL], matrix[ROW * n + k] + matrix[k * n + COL]);
    }
}


void GPU_block_3d_t2_fu(int n, int r, int *matrix) {
    // declare the number of blocks per grid and the number of threads per block
    int nn = n / r;
    dim3 threadsPerBlock(r, r);
    dim3 I_blockPerGrid(1, 1);
    dim3 SD_blockPerGrid(2 * (nn - 1));
    dim3 DD_blockPerGrid((nn - 1),(nn - 1));

    //copy data to gpu
    dev_array<int> d_matrix(n * n);
    d_matrix.set(matrix, n * n);
    for (int nk = 0; nk < nn; nk++) {
        I_block_3d_t2_kernel << < I_blockPerGrid, threadsPerBlock >> > (n, nk * r, (nk + 1) * r, d_matrix.getData());
        SD_block_3d_t2_kernel << < SD_blockPerGrid, threadsPerBlock >> > (n, nk, r, d_matrix.getData());
        DD_block_3d_t2_kernel << < DD_blockPerGrid, threadsPerBlock >> > (n, nk, r, d_matrix.getData());
    }
    d_matrix.get(matrix, n * n);
}